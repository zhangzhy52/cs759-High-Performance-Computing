#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <iostream>

void init_vec(FILE* fp, thrust::host_vector<float>& hv, int n){
	for(int i = 0; i < n; i++){
        int r = fscanf(fp, "%f", &hv[i]);
		if(r == EOF){
			rewind(fp);
		}
	}
}



void inclusive_scan_seq(thrust::host_vector<float>& in, 
                        thrust::host_vector<float>& out,
                        int n) {
	float sum = 0.0f;
	for (int i = 0; i < n; i++) {
		sum += in[i];
		out[i] = sum;
	}
}

int check_results(thrust::host_vector<float> a, 
                  thrust::host_vector<float> b,
                  int n){
    int n_diffs = 0;
    // Keeping this extra high as we have repetitive addition and sequence matters
    const float eps = .3f; 	
    for(int i = 0; i < n; i++)
        if(fabs(a[i]-b[i]) > eps) {
            n_diffs++;
            /* printf("%f %f\n",a[i],b[i]); */
        }
    return n_diffs;
}

int main(int argc, char* argv[]){

    if(argc != 2){
        printf("Usage %s N\n",argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    FILE *fp = fopen("problem1.inp", "r");
    thrust::host_vector<float> h_in(N);
    thrust::host_vector<float> h_out_seq(N);
    thrust::host_vector<float> h_out_thrust(N);
    init_vec(fp, h_in, N);

    thrust::device_vector<float> d_in = h_in;
    thrust::device_vector<float> d_out(N);

    //start inclusive timing
    float time;
    hipEvent_t startIn, stopIn;
    hipEventCreate(&startIn);
    hipEventCreate(&stopIn);
    hipEventRecord(startIn, 0);	

    thrust::inclusive_scan(thrust::device, d_in.begin(), d_in.end(), d_out.begin());

    //stop inclusive timing
    hipEventRecord(stopIn, 0);
    hipEventSynchronize(stopIn);
    hipEventElapsedTime(&time, startIn, stopIn);
    hipEventDestroy(startIn);
    hipEventDestroy(stopIn);

    thrust::copy(d_out.begin(), d_out.end(), h_out_thrust.begin());



    std::cout << N << "\n"
              << h_out_thrust[N-1] << "\n"
              << time << "\n";

    return 0;
}
    


