#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <iostream>

void init_vec(FILE* fp, thrust::host_vector<float>& hv, int n){
	for(int i = 0; i < n; i++){
        int r = fscanf(fp, "%f", &hv[i]);
		if(r == EOF){
			rewind(fp);
		}
	}
}

float reduce_host(thrust::host_vector<float>& in) {
	float sum = 0.0f;
	for (int i = 0; i < in.size(); i++) 
		sum += in[i];
    return sum;
}

int main(int argc, char* argv[]){

    if(argc != 2){
        printf("Usage %s N\n",argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    FILE *fp = fopen("problem1.inp", "r");
    thrust::host_vector<float> h_in(N);
    init_vec(fp, h_in, N);

    thrust::device_vector<float> d_in = h_in;

    //start inclusive timing
    float time;
    hipEvent_t startIn, stopIn;
    hipEventCreate(&startIn);
    hipEventCreate(&stopIn);
    hipEventRecord(startIn, 0);	

    float res_thrust = thrust::reduce(thrust::device, d_in.begin(), d_in.end());

    //stop inclusive timing
    hipEventRecord(stopIn, 0);
    hipEventSynchronize(stopIn);
    hipEventElapsedTime(&time, startIn, stopIn);
    hipEventDestroy(startIn);
    hipEventDestroy(stopIn);


    std::cout << N << "\n"
              << res_thrust << "\n"
              << time << "\n";


    return 0;
}
    


