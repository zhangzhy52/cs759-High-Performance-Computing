#include "hip/hip_runtime.h"
#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <ctime>
// includes, project

// includes, kernels
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define MAX_TILE_SIZE 1024
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

double* read_array(const char* filename, int len) {
	double *x = (double*) malloc(len * sizeof(double));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%lf", &x[i]);
	}
	fclose(fp);
	return x;
}

__global__ void computeOnDevice(double* dA,double* dB, double* dC, int nRows, int tileSize, float* incTime)
{

    __shared__ float ds_M[MAX_TILE_SIZE];
    __shared__ float ds_N[MAX_TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * tileSize + ty,
       Col = bx * tileSize + tx;
    double Pvalue = 0;

    for (int m = 0; m < (nRows-1)/tileSize+1; ++m) {
       if (Row < nRows && m*tileSize+tx < nRows)
          ds_M[ty * tileSize + tx] = dA[Row*nRows + m*tileSize+tx];
       else
          ds_M[ty * tileSize + tx] = 0;
       if (Col < nRows && m*tileSize+ty < nRows)
          ds_N[ty * tileSize + tx] = dB[(m*tileSize+ty)*nRows+Col];
       else
          ds_N[ty * tileSize + tx] = 0;

       __syncthreads();
       for (int k = 0; k < tileSize; ++k)
          Pvalue += ds_M[ty * tileSize + k] * ds_N[k * tileSize + tx];
       __syncthreads();
    }

    if (Row < nRows && Col < nRows)
       dC[Row*nRows+Col] = Pvalue;

	return;//Placeholder
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv) 
{
	if(argc!=2)
	{
		printf("Usage: ./problem2 N\n");
		return 0;
	}
	int nRows = 1024;
	int num_elements = nRows*nRows;
	int tileSize = atoi(argv[1]);  //change this for scaling analysis
	float incTime=0; // Time for GPU
	double* hA = read_array("inputA.inp",num_elements);
	double* hB = read_array("inputB.inp",num_elements);
	double* hC = (double*) malloc(num_elements * sizeof(double));

	dim3 dimGrid((nRows - 1) / tileSize + 1, (nRows - 1) / tileSize + 1, 1);
	dim3 dimBlock(tileSize, tileSize, 1);

	double * dA, *dB, *dC;

	hipError_t error = hipMalloc((void**)&dA, sizeof(double)*num_elements);
	error = hipMalloc((void**)&dB, sizeof(double)*num_elements);
	error = hipMalloc((void**)&dC, sizeof(double)*num_elements);

	hipMemcpy(dA, hA, sizeof(double)*num_elements, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(double)*num_elements, hipMemcpyHostToDevice);

    hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
    hipEventRecord(startEvent_inc,0); // starting timing for inclusive

	// **===-------- Modify the body of this function -----------===**
	computeOnDevice<<<dimGrid, dimBlock>>>(dA, dB, dC, nRows, tileSize, &incTime);
	// **===-----------------------------------------------------------===**
	hipDeviceSynchronize();
	hipMemcpy(hC, dC, sizeof(double)*num_elements, hipMemcpyDeviceToHost);
	
	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&incTime, startEvent_inc, stopEvent_inc);
	printf("%lf\n%f\n%d\n",hC[num_elements - 1],incTime,tileSize);
	// cleanup memory
	free(hA);
	free(hB);
	free(hC);

	return 0;
}


