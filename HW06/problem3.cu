#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vecAddition(double * dA, double *dB, double *dC, int N) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < N){
		dC[id] = dA[id] + dB[id];
	}
}

int main( int argc, char *argv[])
{

	if(argc!=3)
	{
		printf("Invalid argument Usage: ./problem3 N M");
		return 0;
	}

	FILE *fpA,*fpB;
	int N = atoi(argv[1]);
	int M = atoi(argv[2]); 
	double *hA= new double[N];
	double *hB= new double[N];
	double *hC=  new double[N];
	double *refC=  new double[N]; // Used to verify functional correctness
	double *dA,*dB,*dC;  // You may use these to allocate memory on gpu
	//defining variables for timing
	hipEvent_t startEvent_inc, stopEvent_inc, startEvent_exc, stopEvent_exc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventCreate(&startEvent_exc);
	hipEventCreate(&stopEvent_exc);
	float elapsedTime_inc, elapsedTime_exc;

	//reading files
	fpA = fopen("inputA.inp", "r");
	fpB= fopen("inputB.inp", "r");

	int thread_per_block = 32;

	for (int i=0;i<N;i++){    
		fscanf(fpA, "%lf",&hA[i]);
	}
	for (int i=0;i<N;i++){
		fscanf(fpB, "%lf",&hB[i]);
	}

    for(int i=0;i<N;i++)
        refC[i]=hA[i]+hB[i];


	hipEventRecord(startEvent_inc,0); // starting timing for inclusive
	// TODO allocate memory for arrays and copay array A and B

	hipMalloc((void**) &dA, sizeof(double) * N);
	hipMalloc((void**) &dB, sizeof(double) * N);
	hipMalloc((void**) &dC, sizeof(double) * N);
	hipMemcpy(dA, hA, sizeof(double) * N, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(double) * N, hipMemcpyHostToDevice);
	hipEventRecord(startEvent_exc,0); // staring timing for exclusive

	// TODO launch kernel 
	int block_per_grid = N / thread_per_block;

	vecAddition <<< block_per_grid , thread_per_block >>> (dA, dB, dC, N);

	hipDeviceSynchronize();

	hipEventRecord(stopEvent_exc,0);  // ending timing for exclusive
	hipEventSynchronize(stopEvent_exc);   
	hipEventElapsedTime(&elapsedTime_exc, startEvent_exc, stopEvent_exc);

	// TODO copy data back

	hipMemcpy(hC, dC, sizeof(double) * N, hipMemcpyDeviceToHost);

	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);

	//verification
	int count=0;
	for(int i=0;i<N;i++)
	{
		if(hC[i]!=refC[i])
		{
			count++;
		}
	}

	if(count!=0) // This should never be printed in correct code
		std::cout<<"Error at "<< count<<" locations\n";

	std::cout<<N<<"\n"<<M<<"\n";
	std::cout<<elapsedTime_exc<<"\n"<<elapsedTime_inc<<"\n";
	std::cout<<hC[N-1]<<"\n";
	//freeing memory
	delete[] hA,hB,hC,refC;     

	// TODO free CUDA memory allocated

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	return 0;
}