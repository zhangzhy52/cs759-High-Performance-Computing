#include<iostream>

#include<hip/hip_runtime.h>

using namespace std;

__global__ void kernel(int *data)
{
	data[threadIdx.x + blockIdx.x * 8 ] = threadIdx.x + blockIdx.x;
}


int main(){
	const int numElem = 16;
	int hostArray[numElem], *dArray;
	
	// 
	hipMalloc ( (void**) &dArray, sizeof(int) * numElem );
	hipMemset (dArray, 0, numElem * sizeof (int));
	

	kernel <<< 2, 8 >>>( dArray);
	
	hipMemcpy(&hostArray, dArray, sizeof (int) * numElem , hipMemcpyDeviceToHost);

	for (int i = 0 ; i < numElem; i++)
		cout << hostArray[i] << endl;
	hipFree(dArray);
	return 0;
}
