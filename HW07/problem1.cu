#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void multi(int * dA, int *dB, int * dC, int rowWidth, int colWidth){
	//int id = threadIdx.x +  blockIdx.x * blockDim.x;
	int value = 0;
	for (int i = 0; i < blockDim.x; i++)
		value += dA[blockDim.x * blockIdx.x + i] * dB[i];
	dC[blockIdx.x]  = value;
}


int* read_array(const char* filename, int len) {
	int *x = (int*) malloc(len * sizeof(int));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%d", &x[i]);
	}
	fclose(fp);
	return x;
}

int main(int argc, char *argv[]) {
	if (argc != 1) {
		printf("Invalid argument Usage: ./problem1");
		return -1;
	}

	const int rowWidth=32;
    const int colWidth=16;	
	int *hA = read_array("inputA.inp",rowWidth*colWidth );
	int *hB = read_array("inputB.inp", rowWidth);
	int *hC = (int*) malloc(colWidth * sizeof(int));
	int *refC = (int*) malloc (colWidth * sizeof(int));

	// TODO - allocate host memory for refC (you have to figure out how much)
	// The skeleton currently segfaults because refC is accessed without allocation

	// TODO do a reference host implementation (Ch) here. ie populate answer in refC
	for (int i = 0; i < colWidth; i++)
		refC[i] = 0;
	for (int i = 0; i < colWidth; i++)
		for (int j = 0;  j < rowWidth; j++)
		{
			refC[i] += hA[ rowWidth * i + j] * hB[j];
		}



	int *dA, *dB, *dC;
	// TODO allocate device memory for dA,dB and dC
	hipMalloc( (void**) &dA, sizeof(int) * rowWidth * colWidth);
	hipMalloc( (void**) &dB, sizeof(int) * rowWidth);
	hipMalloc((void**) &dC, sizeof(int) * colWidth);


	// TODO copy data from host to GPU 
	hipMemcpy (dA, hA, sizeof(int) * rowWidth * colWidth, hipMemcpyHostToDevice);
	hipMemcpy (dB, hB, sizeof(int) * rowWidth , hipMemcpyHostToDevice);
	hipMemcpy (dC, hC, sizeof(int) * colWidth, hipMemcpyHostToDevice);


	// TODO call your kernel
	multi <<< colWidth ,rowWidth>>> (dA, dB,dC, rowWidth, colWidth);

	// TODO copyback results
	hipMemcpy(hC, dC, sizeof(int) * colWidth, hipMemcpyDeviceToHost);


	int Error=0;

	for(int i=0;i<colWidth;i++)
		Error+=sqrt((hC[i]-refC[i])*(hC[i]-refC[i]));
	printf("%d\n%d",Error,hC[colWidth-1]);

	free(refC);
	free(hB);
	free(hA);

	return 0;
}
