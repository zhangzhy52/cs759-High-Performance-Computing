//#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
//#include <math.h>

#define BLOCK_SIZE 512

int checkResults(float*res, float* cudaRes,int length)
{
	int nDiffs=0;
	const float smallVal = 0.01f; // Keeping this extra high as we have repetitive addition and sequence matters
	for(int i=0; i<length; i++)
		if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
        //if(1)
            //printf("%d :: %f  %f\n", i, cudaRes[i], res[i]);
	return nDiffs;
}

void initializeArray(FILE* fp,float* arr, int nElements)
{
	for( int i=0; i<nElements; i++){
		int r=fscanf(fp,"%f",&arr[i]);
		if(r == EOF){
			rewind(fp);
		}
		arr[i]-=5; // This is to make the data zero mean. Otherwise we reach large numbers and lose precision
	}
}

void inclusiveScan_SEQ(float *in, float *out,int length) {
	float sum=0.f;
	for (int i =0; i < length; i++) {
		sum+=in[i];
		out[i]=sum;
	}
}


__global__ void fixup(float *input, float *aux, int len) {
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
    if (blockIdx.x) {
        if (start + t < len)
            input[start + t] += aux[blockIdx.x - 1];
        if (start + BLOCK_SIZE + t < len)
            input[start + BLOCK_SIZE + t] += aux[blockIdx.x - 1];
    }
}

__global__ void test(float * input, float * output, float *aux, int len) {
        // Load a segment of the input vector into shared memory
        __shared__ float scan_array[BLOCK_SIZE << 1];
            unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
                if (start + t < len)
                           scan_array[t] = input[start + t];
                    else
                               scan_array[t] = 0;
                        if (start + BLOCK_SIZE + t < len)
                                   scan_array[BLOCK_SIZE + t] = input[start + BLOCK_SIZE + t];
                            else
                                       scan_array[BLOCK_SIZE + t] = 0;
                                __syncthreads();

                                    // Reduction
                                    int stride;
                                        for (stride = 1; stride <= BLOCK_SIZE; stride <<= 1) {
                                                   int index = (t + 1) * stride * 2 - 1;
                                                          if (index < 2 * BLOCK_SIZE)
                                                                        scan_array[index] += scan_array[index - stride];
                                                                 __syncthreads();
                                                                     }

                                            // Post reduction
                                            for (stride = BLOCK_SIZE >> 1; stride; stride >>= 1) {
                                                       int index = (t + 1) * stride * 2 - 1;
                                                              if (index + stride < 2 * BLOCK_SIZE)
                                                                            scan_array[index + stride] += scan_array[index];
                                                                     __syncthreads();
                                                                         }

                                                if (start + t < len)
                                                           output[start + t] = scan_array[t];
                                                    if (start + BLOCK_SIZE + t < len)
                                                               output[start + BLOCK_SIZE + t] = scan_array[BLOCK_SIZE + t];

                                                        if (aux && t == 0)
                                                                   aux[blockIdx.x] = scan_array[2 * BLOCK_SIZE - 1];
}

int main(int argc, char* argv[]) {
	if(argc!=2){
		printf("Usage %s N\n",argv[0]);
		return 1;
	}
	int N=atoi(argv[1]);
	FILE *fp = fopen("problem1.inp","r");
	int size = N * sizeof(float); 
	//allocate resources
	float *in      = (float *)malloc(size);
	float *out     = (float *)malloc(size); 
	float *cuda_out= (float *)malloc(size);
	float *d_in, *d_out, * tmp;
    float * deviceAuxScannedArray;
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);
    hipMalloc(&tmp, (BLOCK_SIZE << 1) * sizeof(float));
    hipMalloc(&deviceAuxScannedArray, (BLOCK_SIZE << 1) * sizeof(float));
    float time = 0.f;
	initializeArray(fp,in, N);
	// Your code here
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
    int block_num = N / BLOCK_SIZE;
    if(block_num == 0) block_num++;
    test<<<block_num, BLOCK_SIZE>>>(d_in, d_out, tmp, N);
    hipDeviceSynchronize();
    test<<<dim3(1,1,1), BLOCK_SIZE>>>(tmp, deviceAuxScannedArray, NULL, BLOCK_SIZE << 1);
    hipDeviceSynchronize();
    fixup<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(d_out, deviceAuxScannedArray, N);
    hipDeviceSynchronize();
    hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);
	hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    inclusiveScan_SEQ(in, out,N);
	int nDiffs = checkResults(out, cuda_out,N);

	//if(nDiffs)printf("Test Failed\n"); // This should never print
	printf("%d\n%f\n%f\n",N,cuda_out[N-1],time);



	//free resources 
	free(in); free(out); free(cuda_out);
	return 0;
}